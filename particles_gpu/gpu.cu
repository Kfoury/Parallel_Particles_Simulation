#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"


#define NUM_THREADS 256

extern double size;
// __device__ int find_bin_from_particle(double x_coordinate, double y_coordinate, double size_of_bin, double grid_size_unidirection  );
// __device__  std::vector<int> find_valid_neighbors(int bin_id, int num_bins);
// __global__ void create_bins(int n, int* bins);

//
//  benchmarking program
//
__device__ int find_bin_from_particle(double x_coordinate, double y_coordinate, double size_of_bin, double grid_size_unidirection  )
{
    
    int x_bin;  
    //figuring out the x coordinate
    x_bin = int(x_coordinate / size_of_bin);
    
    int y_bin;
    //figuring out the y coordinate
    y_bin = int(y_coordinate / size_of_bin);
    
    
    return y_bin * (grid_size_unidirection/size_of_bin) + x_bin;
    
}

__device__ void find_valid_neighbors(int bin_id, int num_bins, int* neighbors) {
  int i = 0;

  for (int j = 0; j < 9; j++) {
    neighbors[i] = -1;
  }
  int bin_x, bin_y;
  bin_x = bin_id % num_bins;
  bin_y = int(bin_id / num_bins);
  for (int x = -1; x <= 1; x ++) {
    for (int y = -1; y <=1; y++) {
      if ((x != 0 || y != 0) && bin_x+x >= 0 && bin_x+x < num_bins && bin_y+y >= 0 && bin_y+y < num_bins) {
        neighbors[i] = bin_id+x+y*num_bins;
        i++; 
      }
    }
  }
}

__global__ void create_bins(int n, int* bins) {
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid >= n) return;
  bins[tid] = -1;
}

__global__ void create_particles(int n, int* parts, particle_t* particles, int* bins, double bin_size, int grid_size) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) return;
  int bin = find_bin_from_particle(particles[tid].x, particles[tid].y, bin_size, grid_size);
  parts[tid] = atomicExch(&bins[bin], tid);
}




__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff*cutoff )
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}
__device__ void apply_force(int tid, particle_t* particles, int* parts, int i){
    particle_t* p = &particles[tid];
    while (i != -1)
    {
        apply_force_gpu(*p, particles[i]);
        i = parts[i];
    }
}

__global__ void compute_forces_gpu(particle_t * particles, int* parts, int* bins, int n, int size_of_bin, int grid_size)
{
  // Get thread (particle) ID
  int num_bins = int(grid_size/size_of_bin);
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;
  int bin_id;
  bin_id = find_bin_from_particle(particles[tid].x, particles[tid].y, size_of_bin, grid_size);
  particles[tid].ax = particles[tid].ay = 0;
  int neighbors[9];
  find_valid_neighbors(bin_id, num_bins, neighbors);
  apply_force(tid, particles, parts, bins[bin_id]);
  for (int k = 0; neighbors[k] != -1; k++) {
    apply_force(tid, particles, parts, bins[neighbors[k]]);
  }
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    double size_of_grid = sqrt(n * 0.0005);
    
    //Simple initial guess
    double bin_width = sqrt( 3 * 0.0005);
    int number_of_bins =  int(size_of_grid/bin_width);
    //int number_of_bins =  n % 4 == 0 ? n/4 : n/4 + 1; 
    // int number_of_bins = 16;    
    //Final estimate of the bin width give 3 - 4 particles per bin
    bin_width = size_of_grid / number_of_bins;
    int total_number_of_bins = number_of_bins * number_of_bins;
    int * parts;
    int * bins;
    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int blk2 = (total_number_of_bins + NUM_THREADS - 1) / NUM_THREADS;
    hipMalloc((void **) &parts, n * sizeof(int));
    hipMalloc((void **) &bins, total_number_of_bins * sizeof(int));
    hipDeviceSynchronize();
    create_bins <<< blk2, NUM_THREADS >>> (total_number_of_bins, bins);
    create_particles <<< blks, NUM_THREADS >>> (n, parts, d_particles, bins, bin_width, size_of_grid);

    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

      compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, parts, bins, n, bin_width, size_of_grid);
        
        //
        //  move particles
        //
      move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
      
      create_bins <<< blk2, NUM_THREADS >>> (total_number_of_bins, bins);
      create_particles <<< blks, NUM_THREADS >>> (n, parts, d_particles, bins, bin_width, size_of_grid);

        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
      // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    hipFree(parts);
    hipFree(bins);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
